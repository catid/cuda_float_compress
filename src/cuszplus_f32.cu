#include "hip/hip_runtime.h"
#include "cuszplus_f32.h"

#include <stdio.h>

__device__ inline int quantization_f32(float data, float recipPrecision)
{
    float dataRecip = data*recipPrecision;
    int s = dataRecip>=-0.5f?0:1;
    return (int)(dataRecip+0.5f) - s;
}

__device__ inline int get_bit_num(unsigned int x)
{
    return (sizeof(unsigned int)*8) - __clz(x);
}

__global__ void SZplus_compress_kernel_f32(
    const float* const __restrict__ oriData,
    unsigned char* const __restrict__ cmpData,
    volatile unsigned int* const __restrict__ cmpOffset,
    volatile int* const __restrict__ flag,
    const float eb,
    const size_t nbEle)
{
    __shared__ unsigned int base_idx;

    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int lane = idx & 31;
    const int warp = idx >> 5;
    const int block_num = cmp_chunk_f32/32;
    const int start_idx = idx * cmp_chunk_f32;
    const int start_block_idx = start_idx/32;
    const int rate_ofs = (nbEle+31)/32;
    const float recipPrecision = 0.5f/eb;

    int temp_start_idx, temp_end_idx;
    int quant_chunk_idx;
    int block_idx;
    int currQuant, lorenQuant, prevQuant, maxQuant;
    int absQuant[cmp_chunk_f32];
    unsigned int sign_flag[block_num];
    int sign_ofs;
    int fixed_rate[block_num];
    unsigned int thread_ofs = 0;

    for(int j=0; j<block_num; j++)
    {
        sign_flag[j] = 0;
        temp_start_idx = start_idx + j*32;
        temp_end_idx = temp_start_idx + 32;
        block_idx = start_block_idx+j;
        prevQuant = 0;
        maxQuant = 0;

        for(int i=temp_start_idx; i<temp_end_idx; i++)
        {
            quant_chunk_idx = i%cmp_chunk_f32;
            currQuant = quantization_f32(oriData[i], recipPrecision);
            lorenQuant = currQuant - prevQuant;
            prevQuant = currQuant;
            sign_ofs = i % 32;
            sign_flag[j] |= (lorenQuant < 0) << (31 - sign_ofs);
            absQuant[quant_chunk_idx] = abs(lorenQuant);
            maxQuant = maxQuant > absQuant[quant_chunk_idx] ? maxQuant : absQuant[quant_chunk_idx];
        }

        fixed_rate[j] = get_bit_num(maxQuant);
        thread_ofs += (fixed_rate[j]) ? (32+fixed_rate[j]*32) : 0;
        if(block_idx<rate_ofs) cmpData[block_idx] = (unsigned char)fixed_rate[j];
    }
    __syncthreads();

    for(int i=1; i<32; i<<=1)
    {
        int tmp = __shfl_up_sync(0xffffffff, thread_ofs, i);
        if(lane >= i) thread_ofs += tmp;
    }
    __syncthreads();

    if(lane==31) 
    {
        cmpOffset[warp+1] = (thread_ofs+7)/8;
        __threadfence();
        if(warp==0)
        {
            flag[1] = 2;
            __threadfence();
        }
        else
        {
            flag[warp+1] = 1;
            __threadfence();
        }
    }
    __syncthreads();

    if(warp>0)
    {
        if(!lane)
        {
            int temp_flag = 1;
            while(temp_flag!=2) temp_flag = flag[warp];
            __threadfence();
            cmpOffset[warp] += cmpOffset[warp-1];
            if(warp==gridDim.x-1) cmpOffset[warp+1] += cmpOffset[warp];
            __threadfence();
            flag[warp+1] = 2;
        }
        
    }
    __syncthreads();

    if(!lane) base_idx = cmpOffset[warp] + rate_ofs;
    __syncthreads();

    unsigned int prev_thread = __shfl_up_sync(0xffffffff, thread_ofs, 1);
    unsigned int cmp_byte_ofs;
    if(!lane) cmp_byte_ofs = base_idx;
    else cmp_byte_ofs = base_idx + prev_thread / 8;
    
    for(int j=0; j<block_num; j++)  
    {
        int chunk_idx_start = j*32;
        int rate = fixed_rate[j];

        if(rate != 0)
        {
            cmpData[cmp_byte_ofs++] = 0xff & (sign_flag[j] >> 24);
            cmpData[cmp_byte_ofs++] = 0xff & (sign_flag[j] >> 16);
            cmpData[cmp_byte_ofs++] = 0xff & (sign_flag[j] >> 8);
            cmpData[cmp_byte_ofs++] = 0xff & sign_flag[j];

            unsigned char tmp_char0, tmp_char1, tmp_char2, tmp_char3;
            int mask = 1;
            for(int i=0; i<rate; i++)
            {
                tmp_char0 = 0;
                tmp_char1 = 0;
                tmp_char2 = 0;
                tmp_char3 = 0;

                tmp_char0 = (((absQuant[chunk_idx_start+0] & mask) >> i) << 7) |
                            (((absQuant[chunk_idx_start+1] & mask) >> i) << 6) |
                            (((absQuant[chunk_idx_start+2] & mask) >> i) << 5) |
                            (((absQuant[chunk_idx_start+3] & mask) >> i) << 4) |
                            (((absQuant[chunk_idx_start+4] & mask) >> i) << 3) |
                            (((absQuant[chunk_idx_start+5] & mask) >> i) << 2) |
                            (((absQuant[chunk_idx_start+6] & mask) >> i) << 1) |
                            (((absQuant[chunk_idx_start+7] & mask) >> i) << 0);

                tmp_char1 = (((absQuant[chunk_idx_start+8] & mask) >> i) << 7) |
                            (((absQuant[chunk_idx_start+9] & mask) >> i) << 6) |
                            (((absQuant[chunk_idx_start+10] & mask) >> i) << 5) |
                            (((absQuant[chunk_idx_start+11] & mask) >> i) << 4) |
                            (((absQuant[chunk_idx_start+12] & mask) >> i) << 3) |
                            (((absQuant[chunk_idx_start+13] & mask) >> i) << 2) |
                            (((absQuant[chunk_idx_start+14] & mask) >> i) << 1) |
                            (((absQuant[chunk_idx_start+15] & mask) >> i) << 0);

                tmp_char2 = (((absQuant[chunk_idx_start+16] & mask) >> i) << 7) |
                            (((absQuant[chunk_idx_start+17] & mask) >> i) << 6) |
                            (((absQuant[chunk_idx_start+18] & mask) >> i) << 5) |
                            (((absQuant[chunk_idx_start+19] & mask) >> i) << 4) |
                            (((absQuant[chunk_idx_start+20] & mask) >> i) << 3) |
                            (((absQuant[chunk_idx_start+21] & mask) >> i) << 2) |
                            (((absQuant[chunk_idx_start+22] & mask) >> i) << 1) |
                            (((absQuant[chunk_idx_start+23] & mask) >> i) << 0);
                
                tmp_char3 = (((absQuant[chunk_idx_start+24] & mask) >> i) << 7) |
                            (((absQuant[chunk_idx_start+25] & mask) >> i) << 6) |
                            (((absQuant[chunk_idx_start+26] & mask) >> i) << 5) |
                            (((absQuant[chunk_idx_start+27] & mask) >> i) << 4) |
                            (((absQuant[chunk_idx_start+28] & mask) >> i) << 3) |
                            (((absQuant[chunk_idx_start+29] & mask) >> i) << 2) |
                            (((absQuant[chunk_idx_start+30] & mask) >> i) << 1) |
                            (((absQuant[chunk_idx_start+31] & mask) >> i) << 0);

                // Move data to global memory.
                cmpData[cmp_byte_ofs++] = tmp_char0;
                cmpData[cmp_byte_ofs++] = tmp_char1;
                cmpData[cmp_byte_ofs++] = tmp_char2;
                cmpData[cmp_byte_ofs++] = tmp_char3;
                mask <<= 1;
            }
        }
    }
}

__global__ void SZplus_decompress_kernel_f32(
    float* const __restrict__ decData,
    const unsigned char* const __restrict__ cmpData,
    volatile unsigned int* const __restrict__ cmpOffset,
    volatile int* const __restrict__ flag,
    const float eb,
    const size_t nbEle)
{
    __shared__ unsigned int base_idx;

    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int lane = idx & 31;
    const int warp = idx >> 5;
    const int block_num = dec_chunk_f32/32;
    const int start_idx = idx * dec_chunk_f32;
    const int start_block_idx = start_idx/32;
    const int rate_ofs = (nbEle+31)/32;

    int temp_start_idx;
    int block_idx;
    int absQuant[32];
    int currQuant, lorenQuant, prevQuant;
    int sign_ofs;
    int fixed_rate[block_num];
    unsigned int thread_ofs = 0;

    for(int j=0; j<block_num; j++)
    {
        block_idx = start_block_idx + j;
        if(block_idx<rate_ofs) 
        {
            fixed_rate[j] = (int)cmpData[block_idx];
            thread_ofs += (fixed_rate[j]) ? (32+fixed_rate[j]*32) : 0;
        }
    }
    __syncthreads();

    for(int i=1; i<32; i<<=1)
    {
        int tmp = __shfl_up_sync(0xffffffff, thread_ofs, i);
        if(lane >= i) thread_ofs += tmp;
    }
    __syncthreads();

    if(lane==31) 
    {
        cmpOffset[warp+1] = (thread_ofs+7)/8;
        __threadfence();
        if(warp==0)
        {
            flag[1] = 2;
            __threadfence();
        }
        else
        {
            flag[warp+1] = 1;
            __threadfence();
        }
    }
    __syncthreads();

    if(warp>0)
    {
        if(!lane)
        {
            int temp_flag = 1;
            while(temp_flag!=2) temp_flag = flag[warp];
            __threadfence();
            cmpOffset[warp] += cmpOffset[warp-1];
            __threadfence();
            flag[warp+1] = 2;
        }
    }
    __syncthreads();

    if(!lane) base_idx = cmpOffset[warp] + rate_ofs;
    __syncthreads();

    unsigned int prev_thread = __shfl_up_sync(0xffffffff, thread_ofs, 1);
    unsigned int cmp_byte_ofs;
    if(!lane) cmp_byte_ofs = base_idx;
    else cmp_byte_ofs = base_idx + prev_thread / 8;

    for(int j=0; j<block_num; j++)
    {
        temp_start_idx = start_idx + j*32;
        unsigned int sign_flag = 0;

        if(fixed_rate[j])
        {
            sign_flag = (0xff000000 & (cmpData[cmp_byte_ofs++] << 24)) |
                        (0x00ff0000 & (cmpData[cmp_byte_ofs++] << 16)) |
                        (0x0000ff00 & (cmpData[cmp_byte_ofs++] << 8))  |
                        (0x000000ff & cmpData[cmp_byte_ofs++]);
            
            unsigned char tmp_char0, tmp_char1, tmp_char2, tmp_char3;
            for(int i=0; i<32; i++) absQuant[i] = 0;
            for(int i=0; i<fixed_rate[j]; i++)
            {
                tmp_char0 = cmpData[cmp_byte_ofs++];
                tmp_char1 = cmpData[cmp_byte_ofs++];
                tmp_char2 = cmpData[cmp_byte_ofs++];
                tmp_char3 = cmpData[cmp_byte_ofs++];

                absQuant[0] |= ((tmp_char0 >> 7) & 0x00000001) << i;
                absQuant[1] |= ((tmp_char0 >> 6) & 0x00000001) << i;
                absQuant[2] |= ((tmp_char0 >> 5) & 0x00000001) << i;
                absQuant[3] |= ((tmp_char0 >> 4) & 0x00000001) << i;
                absQuant[4] |= ((tmp_char0 >> 3) & 0x00000001) << i;
                absQuant[5] |= ((tmp_char0 >> 2) & 0x00000001) << i;
                absQuant[6] |= ((tmp_char0 >> 1) & 0x00000001) << i;
                absQuant[7] |= ((tmp_char0 >> 0) & 0x00000001) << i;

                absQuant[8] |= ((tmp_char1 >> 7) & 0x00000001) << i;
                absQuant[9] |= ((tmp_char1 >> 6) & 0x00000001) << i;
                absQuant[10] |= ((tmp_char1 >> 5) & 0x00000001) << i;
                absQuant[11] |= ((tmp_char1 >> 4) & 0x00000001) << i;
                absQuant[12] |= ((tmp_char1 >> 3) & 0x00000001) << i;
                absQuant[13] |= ((tmp_char1 >> 2) & 0x00000001) << i;
                absQuant[14] |= ((tmp_char1 >> 1) & 0x00000001) << i;
                absQuant[15] |= ((tmp_char1 >> 0) & 0x00000001) << i;

                absQuant[16] |= ((tmp_char2 >> 7) & 0x00000001) << i;
                absQuant[17] |= ((tmp_char2 >> 6) & 0x00000001) << i;
                absQuant[18] |= ((tmp_char2 >> 5) & 0x00000001) << i;
                absQuant[19] |= ((tmp_char2 >> 4) & 0x00000001) << i;
                absQuant[20] |= ((tmp_char2 >> 3) & 0x00000001) << i;
                absQuant[21] |= ((tmp_char2 >> 2) & 0x00000001) << i;
                absQuant[22] |= ((tmp_char2 >> 1) & 0x00000001) << i;
                absQuant[23] |= ((tmp_char2 >> 0) & 0x00000001) << i;

                absQuant[24] |= ((tmp_char3 >> 7) & 0x00000001) << i;
                absQuant[25] |= ((tmp_char3 >> 6) & 0x00000001) << i;
                absQuant[26] |= ((tmp_char3 >> 5) & 0x00000001) << i;
                absQuant[27] |= ((tmp_char3 >> 4) & 0x00000001) << i;
                absQuant[28] |= ((tmp_char3 >> 3) & 0x00000001) << i;
                absQuant[29] |= ((tmp_char3 >> 2) & 0x00000001) << i;
                absQuant[30] |= ((tmp_char3 >> 1) & 0x00000001) << i;
                absQuant[31] |= ((tmp_char3 >> 0) & 0x00000001) << i;
            }
            prevQuant = 0;
            for(int i=0; i<32; i++)
            {
                sign_ofs = i % 32;
                if(sign_flag & (1 << (31 - sign_ofs)))
                    lorenQuant = absQuant[i] * -1;
                else
                    lorenQuant = absQuant[i];
                currQuant = (lorenQuant + prevQuant) * 2;
                decData[temp_start_idx+i] = currQuant * eb;
                prevQuant = currQuant;
            }
        } else {
            for(int i=0; i<32; i++)
            {
                decData[temp_start_idx+i] = 0.f;
            }
        }
    }

    printf("cmp_byte_ofs = %d\n", (int)cmp_byte_ofs);
}

int SZplus_compress_hostptr_f32(
    float* oriData,
    unsigned char* cmpBytes,
    size_t nbEle,
    size_t* cmpSize,
    float errorBound)
{
    // Data blocking.
    int bsize = cmp_tblock_size_f32;
    int gsize = (nbEle + bsize * cmp_chunk_f32 - 1) / (bsize * cmp_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f32 * 2;

    // Initializing global memory for GPU compression.
    float* d_oriData = NULL;
    unsigned char* d_cmpData = NULL;
    unsigned int* d_cmpOffset = NULL;
    int* d_flag = NULL;
    hipError_t err = hipMalloc((void**)&d_oriData, sizeof(float)*pad_nbEle);
    printf("pad_nbEle: %zu\n", pad_nbEle);
    printf("d_oriData: %p\n", d_oriData);
    printf("err = %d\n", err);
    printf("gsize: %zu\n", gsize);
    printf("bsize: %zu\n", bsize);
    printf("cmp_chunk_f32: %zu\n", cmp_chunk_f32);
    printf("nbEle: %zu\n", nbEle);
    if (err != hipSuccess) { return -1; }

    hipMemcpy(d_oriData, oriData, sizeof(float)*nbEle, hipMemcpyHostToDevice);
    printf("pad_nbEle: %zu\n", pad_nbEle);
    err = hipMalloc((void**)&d_cmpData, sizeof(float)*pad_nbEle);
    if (err != hipSuccess) { return -1; }

    printf("cmpOffSize: %zu\n", cmpOffSize);
    err = hipMallocManaged((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    if (err != hipSuccess) { return -1; }

    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    err = hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    printf("cmpOffSize: %zu\n", cmpOffSize);
    if (err != hipSuccess) { return -1; }

    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);
    hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(float));

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZplus_compress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_cmpData, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU.  
    printf("nbEle % 32 = %d\n", (int)(nbEle % 32));
    *cmpSize = (size_t)d_cmpOffset[cmpOffSize-1] + (nbEle+31)/32;
    hipMemcpy(cmpBytes, d_cmpData, *cmpSize, hipMemcpyDeviceToHost);

    printf("sizeof(float)*pad_nbEle = %zu\n", sizeof(float)*pad_nbEle);
    printf("*cmpSize = %d\n", (int)*cmpSize);

    // Free memory that is used.
    hipFree(d_oriData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);

    return 0;
}

int SZplus_decompress_hostptr_f32(
    float* decData,
    unsigned char* cmpBytes,
    size_t nbEle,
    size_t cmpSize,
    float errorBound)
{
    // Data blocking.
    int bsize = dec_tblock_size_f32;
    int gsize = (nbEle + bsize * dec_chunk_f32 - 1) / (bsize * dec_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * dec_chunk_f32;

    // Initializing global memory for GPU compression.
    float* d_decData = NULL;
    unsigned char* d_cmpData = NULL;
    unsigned int* d_cmpOffset = NULL;
    int* d_flag = NULL;
    hipError_t err = hipMalloc((void**)&d_decData, sizeof(float)*pad_nbEle);
    if (err != hipSuccess) { return -1; }
    //hipMemset(d_decData, 0, sizeof(float)*pad_nbEle);
    err = hipMalloc((void**)&d_cmpData, sizeof(float)*pad_nbEle);
    if (err != hipSuccess) { return -1; }
    hipMemcpy(d_cmpData, cmpBytes, cmpSize, hipMemcpyHostToDevice);
    err = hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    if (err != hipSuccess) { return -1; }
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    err = hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    if (err != hipSuccess) { return -1; }
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    printf("gridSize.x: %d\n", gridSize.x);
    printf("blockSize.x: %d\n", blockSize.x);
    SZplus_decompress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_decData, d_cmpData, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Move data back to CPU.
    hipMemcpy(decData, d_decData, sizeof(float)*nbEle, hipMemcpyDeviceToHost);

    // Free memoy that is used.
    hipFree(d_decData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);

    return 0;
}

int SZplus_compress_deviceptr_f32(
    float* d_oriData,
    unsigned char* d_cmpBytes,
    size_t nbEle,
    size_t* cmpSize,
    float errorBound,
    hipStream_t stream)
{
    // Data blocking.
    int bsize = cmp_tblock_size_f32;
    int gsize = (nbEle + bsize * cmp_chunk_f32 - 1) / (bsize * cmp_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f32;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset = NULL;
    int* d_flag = NULL;
    hipError_t err = hipMallocManaged((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    if (err != hipSuccess) { return -1; }
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    err = hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    if (err != hipSuccess) { return -1; }
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);
    hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(float));

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZplus_compress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU.  
    *cmpSize = (size_t)d_cmpOffset[cmpOffSize-1] + (nbEle+31)/32;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_flag);

    return 0;
}

int SZplus_decompress_deviceptr_f32(
    float* d_decData,
    unsigned char* d_cmpBytes,
    size_t nbEle,
    size_t cmpSize,
    float errorBound,
    hipStream_t stream)
{
    // Data blocking.
    int bsize = dec_tblock_size_f32;
    int gsize = (nbEle + bsize * dec_chunk_f32 - 1) / (bsize * dec_chunk_f32);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset = NULL;
    int* d_flag = NULL;
    hipError_t err = hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    if (err != hipSuccess) { return -1; }
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    err = hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    if (err != hipSuccess) { return -1; }
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZplus_decompress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Free memoy that is used.
    hipFree(d_cmpOffset);
    hipFree(d_flag);

    return 0;
}
